////////////////////////////////////////////////////////////////////////////////
// utilities
////////////////////////////////////////////////////////////////////////////////


#include <hip/hip_runtime.h>
#include <limits.h>  // INT_MIN
#include <math.h>    // sqrt
#include <stdlib.h>  // abs, atoi
#include <sys/time.h>
#include <time.h>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace std;

typedef unsigned short int usint;
typedef unsigned long long int ullint;

void die(const char *msg) {
  cerr << "ERROR: " << msg << endl;
  exit(1);
}

#define RANDOM_MAX 0x7fffffff  // 2^31 - 1

__device__ int random(const int seed) {
  // glibc: m = 2^31; a = 1103515245; c = 12345
  return (int)((1103515245U * ((unsigned)seed & 0x7fffffffU) + 12345U) &
               0x7fffffffU);
}

template <class C>
__device__ C sign(const C x) {
  return x < 0 ? -1 : (x == 0 ? 0 : 1);
}

template <class C>
__host__ __device__ C square(const C x) {
  return x * x;
}

inline double calc_time(struct timeval &begin, struct timeval &end) {
  return ((end.tv_sec - begin.tv_sec) * 1000000u +
          end.tv_usec - begin.tv_usec) / 1e6;
}

////////////////////////////////////////////////////////////////////////////////
// vix
////////////////////////////////////////////////////////////////////////////////

__device__ int test_one_target(const int nrows, const usint *elevs,
                               const int ox, const int oy, const int oz,
                               const int tx, const int ty, const int tz) {
  if (abs(ox - tx) <= 1 && abs(oy - ty) <= 1) return 1;
  int dx = tx - ox, dy = ty - oy;
  int px, py, pz;  // Current point
  int inciny = abs(dx) < abs(dy);
  int sign;
  float slope, zslope;
  sign = (inciny*dy + (1-inciny)*dx) > 0 ? 1 : -1;
  slope = (float)(inciny*dx + (1-inciny)*dy) / (inciny*dy + (1-inciny)*dx);
  zslope = (float)(tz - oz) / (inciny ? dy : dx);
  const int limit = inciny ? dy : dx;
  int stride = 1;
  for (int i = sign; abs(i) < abs(limit); i += stride*sign, stride <<= 1) {  // *= 1.9, 2.2, 2.5
    int j = round(i * slope);
    px = ox + (inciny*j + (1-inciny)*i);
    py = oy + (inciny*i + (1-inciny)*j);
    pz = elevs[px * nrows + py];
    if (pz > oz + i * zslope) return 0;
  }
  return 1;
}

__global__ void calc_one_vix(const int nrows, const usint *elevs,
                             const int roi, const int oht, const int tht,
                             const int ntests, unsigned char *vix) {
  const int bid = blockIdx.y * gridDim.x + blockIdx.x;
  const int tid = bid * blockDim.x + threadIdx.x;
  if (tid >= square(nrows)) return;

  const int ox = tid / nrows;
  const int oy = tid % nrows;
  const int oz = elevs[ox * nrows + oy] + oht;
  int ntarget = 0;
  int nvis = 0;
  const int vsxmax = min(nrows - 1, ox + roi);  // viewshed bounds
  const int vsymax = min(nrows - 1, oy + roi);
  const int vsxmin = max(0, ox - roi);
  const int vsymin = max(0, oy - roi);
  int r = tid;
  // for (int i = 0; i < 10 * ntests; i++) {  // iterate over random targets
  while (ntarget < ntests) {
    int tx, ty, tz;
    int visq;
    r = random(r);
    //tx = (int)(r * (vsxmax - vsxmin + 0.99999f) / RANDOM_MAX) + vsxmin;
    tx = (int)((2*roi+0.99999f)*r/RANDOM_MAX) + (ox-roi);
    r = random(r);
    //ty = (int)(r * (vsymax - vsymin + 0.99999f) / RANDOM_MAX) + vsymin;
    ty = (int)((2*roi+0.99999f)*r/RANDOM_MAX) + (oy-roi);
    // if (tx == 0 && ty == 0) continue;
    if (square(tx - ox) + square(ty - oy) > square(roi)) {
      tx = ox + (tx - ox)/3; // golden
      ty = oy + (ty - oy)/3;
      //continue; // too slow
    }
    if (tx >=0 && tx < nrows && ty >= 0 && ty < nrows) {
        tz = elevs[tx * nrows + ty] + tht;
        visq = test_one_target(nrows, elevs, ox, oy, oz, tx, ty, tz);
        // cerr << "test_one_target(" << ox << ',' << oy << ',' << tx << ',' << ty << ")=" << visq << endl;
    } else {
        visq = 0;
    }
    ntarget++;
    if (visq) nvis++;

    // Stopping rule: do at least 10 points.
    // Then, continue until vix >= .5 or <= .1.
    // This could be improved by using the variance, and by looking at
    // other observers, to select the best.

    // if (ntarget >= ntests) break;
    // if (ntarget < 10) continue;
    // if (v >= 0.5 || v <= 0.1) break;
  }
  float v = (float)nvis / ntarget;
  // cerr << "obs at (" << ox << ',' << oy << "), z=" << elevs[ox * nrows + oy] 
  //      << ", vix=" << nvis << '/' << ntarget << '=' << v << endl;
  vix[ox * nrows + oy] = (unsigned char)min(255, (int)(v * 255.999f));
}

void calc_vix(const int nrows, const usint *h_elevs,
              const int roi, const int oht, const int tht,
              const int ntests, unsigned char *h_vix) {
  usint *d_elevs;
  unsigned char *d_vix;

  if (hipMalloc((void **)&d_elevs, square(nrows) * sizeof(usint)) != hipSuccess)
    die("hipMalloc failed");
  if (hipMalloc((void **)&d_vix, square(nrows) * sizeof(unsigned char)) != hipSuccess)
    die("hipMalloc failed");
  if (hipMemcpy(d_elevs, h_elevs, square(nrows) * sizeof(usint), hipMemcpyHostToDevice) != hipSuccess)
    die("hipMemcpy failed");

  const size_t dimblock = 128;
  // const size_t dimgrid = square(nrows) / dimblock + (square(nrows) % dimblock ? 1 : 0);
  int s = (int)sqrt(square(nrows) / dimblock);
  if (square(s) * dimblock < square(nrows)) s++;
  const dim3 dimgrid(s, s);
  calc_one_vix<<<dimgrid, dimblock>>>(nrows, d_elevs, roi, oht, tht, ntests, d_vix);

  if (hipMemcpy(h_vix, d_vix, square(nrows) * sizeof(unsigned char), hipMemcpyDeviceToHost) != hipSuccess)
    die("hipMemcpy failed");

  if (hipFree(d_elevs) != hipSuccess) die("hipFree failed");
  if (hipFree(d_vix) != hipSuccess) die("hipFree failed");
}

////////////////////////////////////////////////////////////////////////////////
// findmax
////////////////////////////////////////////////////////////////////////////////

__global__ void process_one_block(const int nrows, unsigned char *vix,
                                  const float blocksize, const int nblockrows,
                                  const int nwantedperblock, int *obs) {
  // no need for if (blockIdx.x >= square(nblockrows)) return;
  extern __shared__ int results[];
  __shared__ int xmin, xmax, ymin, ymax;
  if (threadIdx.x == 0) {
    int bx = blockIdx.x / nblockrows;
    int by = blockIdx.x % nblockrows;
    xmin = (int)(blocksize * bx);
    xmax = min((int)(blocksize * (bx + 1)), nrows);
    ymin = (int)(blocksize * by);
    ymax = min((int)(blocksize * (by + 1)), nrows);
  }
  __syncthreads();

  const int width = ymax - ymin;
  const int npoints = (xmax - xmin) * (ymax - ymin);
  const int npointsperthread = npoints / blockDim.x + (npoints % blockDim.x ? 1 : 0);
  for (int i = 0; i < nwantedperblock; i++) {
    int t = threadIdx.x * npointsperthread;  // the first point, probably used
    int p1x = xmin + t / width;
    int p1y = ymin + t % width;
    unsigned char v1 = vix[p1x * nrows + p1y];
    int h1 = p1x * (p1x + p1y) * 010101010101;
    for (int j = t + 1; j < t + npointsperthread && j < npoints; j++) {
      int p2x = xmin + j / width;
      int p2y = ymin + j % width;
      unsigned char v2 = vix[p2x * nrows + p2y];
      int h2 = p2x * (p2x + p2y) * 010101010101;
      if (v1 < v2 || (v1 == v2 && h1 < h2)) {
        p1x = p2x;
        p1y = p2y;
        v1 = v2;
        h1 = h2;
      }
    }
    results[threadIdx.x * 4] = p1x;
    results[threadIdx.x * 4 + 1] = p1y;
    results[threadIdx.x * 4 + 2] = v1;
    results[threadIdx.x * 4 + 3] = h1;
    if (threadIdx.x == 0) {
      // p1x... is the result of thread 0
      for (int j = 1; j < blockDim.x; j++) {
        int p2x = results[j * 4];
        int p2y = results[j * 4 + 1];
        int v2 = results[j * 4 + 2];
        int h2 = results[j * 4 + 3];
        if (v1 < v2 || (v1 == v2 && h1 < h2)) {
          p1x = p2x;
          p1y = p2y;
          v1 = v2;
          h1 = h2;
        }
      }
      obs[blockIdx.x * nwantedperblock * 2 + i * 2] = p1x;
      obs[blockIdx.x * nwantedperblock * 2 + i * 2 + 1] = p1y;
      vix[p1x * nrows + p1y] = 0;  // used
    }
    __syncthreads();
  }
}

void find_max(const int nrows, const unsigned char *h_vix,
              const float blocksize, const int nwanted, const int nblockrows,
              const int nwantedperblock, int *h_obs) {
  unsigned char *d_vix;
  int *d_obs;
  if (hipMalloc((void **)&d_vix, square(nrows) * sizeof(unsigned char)) != hipSuccess)
    die("hipMalloc failed");
  if (hipMalloc((void **)&d_obs, nwanted * 2 * sizeof(int)) != hipSuccess)
    die("hipMalloc failed");
  if (hipMemcpy(d_vix, h_vix, square(nrows) * sizeof(unsigned char), hipMemcpyHostToDevice) != hipSuccess)
    die("hipMemcpy failed");

  const size_t dimgrid = square(nblockrows);  // two dimensional
  const size_t dimblock = 256;
  process_one_block<<<dimgrid, dimblock, dimblock * 4 * sizeof(int)>>>(
      nrows, d_vix, blocksize, nblockrows, nwantedperblock, d_obs);

  if (hipMemcpy(h_obs, d_obs, nwanted * 2 * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
    die("hipMemcpy failed");

  if (hipFree(d_vix) != hipSuccess) die("hipFree failed");
  if (hipFree(d_obs) != hipSuccess) die("hipFree failed");
}

////////////////////////////////////////////////////////////////////////////////
// viewshed
////////////////////////////////////////////////////////////////////////////////

__device__ void set_vis(const int nwpr, const int row, const int col,
                        ullint *shed) {
  atomicOr(&shed[row * nwpr + col / 64], 1ULL << (63 - col % 64));
}

__global__ void calc_one_shed(const int nrows, const usint *elevs,
                              const int roi, const int oht, const int tht,
                              const int nsheds, const int *obs,
                              ullint *sheds) {
  if (blockIdx.x >= nsheds) return;
  const int nr = 2 * roi + 1;
  const int nwpr = (nr + 63) / 64;
  ullint * const thisshed = &sheds[blockIdx.x * nr * nwpr];
  __shared__ int ox, oy, oz;
  if (threadIdx.x == 0) {
    ox = obs[2 * blockIdx.x];
    oy = obs[2 * blockIdx.x + 1];
    oz = elevs[ox * nrows + oy] + oht;
    set_vis(nwpr, roi, roi, thisshed);
  }
  __syncthreads();

  // Clipping xmin etc at 0, nrows-1 makes the viewshed depend on the roi, so don't.
  const int xmin = ox - roi;
  const int ymin = oy - roi;
  const int xmax = ox + roi;
  const int ymax = oy + roi;
  const int xwidth = xmax - xmin;
  const int ywidth = ymax - ymin;
  const int perimeter = 2 * (xwidth + ywidth);  // This formula is subtle
  const int ntpt = perimeter / blockDim.x + (perimeter % blockDim.x ? 1 : 0);
  int dx, dy;
  int tx, ty;
  int px, py;  // Current point
  int inciny;
  int sign;
  float slope, zslope;

  const int sector = threadIdx.x;
  for (int ip = sector * ntpt; ip < (sector + 1) * ntpt && ip < perimeter; ip++) {
    if (ip < xwidth) {
      tx = xmin + ip;
      ty = ymin;
    } else if (ip < 2 * xwidth) {
      tx = 1 + xmin - xwidth + ip;
      ty = ymax;
    } else if (ip < 2 * xwidth + ywidth) {
      tx = xmin;
      ty = 1 + ymin - 2 * xwidth + ip;
    } else {
      tx = xmax;
      ty = ymin - 2 * xwidth - ywidth + ip;
    }

    // Run a line of sight out from obs to target.
    dx = tx - ox;
    dy = ty - oy;
    inciny = abs(dx) < abs(dy);
    sign = (inciny*dy + (1-inciny)*dx) > 0 ? 1 : -1;
    slope = (float)(inciny*dx + (1-inciny)*dy) / (inciny*dy + (1-inciny)*dx);
    zslope = -99999.f;

    // i=0 would be the observer, which is always visible.
    for (int i = sign; i != (inciny ? dy : dx) + sign; i += sign) {
      int j = round(i * slope);
      px = ox + (inciny*j + (1-inciny)*i);
      py = oy + (inciny*i + (1-inciny)*j);

      // Have we reached the edge of the area?
      if (px < 0 || px >= nrows || py < 0 || py >= nrows) break;
      if (square(px - ox) + square(py - oy) > square(roi)) break;

      int pelev = elevs[px * nrows + py];
      float s = (float)(pelev - oz) / abs(i);
      if (zslope < s) zslope = s;
      float hz = oz + zslope * abs(i);
      if (pelev + tht >= hz)
        set_vis(nwpr, px - ox + roi, py - oy + roi, thisshed);
    }
  }
}

void calc_sheds(const int nrows, const usint *h_elevs,
                const int roi, const int oht, const int tht,
                const int nsheds, const int *h_obs,
                ullint *h_sheds) {
  usint *d_elevs;
  int *d_obs;
  ullint *d_sheds;
  const int nr = 2 * roi + 1;
  const int nwpr = (nr + 63) / 64;

  if (hipMalloc((void **)&d_elevs, square(nrows) * sizeof(usint)) != hipSuccess)
    die("hipMalloc failed");
  if (hipMalloc((void **)&d_obs, 2 * nsheds * sizeof(int)) != hipSuccess)
    die("hipMalloc failed");
  if (hipMalloc((void **)&d_sheds, nsheds * nr * nwpr * sizeof(ullint)) != hipSuccess)
    die("hipMalloc failed");
  if (hipMemcpy(d_elevs, h_elevs, square(nrows) * sizeof(usint), hipMemcpyHostToDevice) != hipSuccess)
    die("hipMemcpy failed");
  if (hipMemcpy(d_obs, h_obs, 2 * nsheds * sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
    die("hipMemcpy failed");
  if (hipMemset(d_sheds, 0, nsheds * nr * nwpr * sizeof(ullint)) != hipSuccess)
    die("hipMemset failed");

  const size_t dimblock = 256;  // must be a multiple of 32
  const size_t dimgrid = nsheds;
  calc_one_shed<<<dimgrid, dimblock>>>(nrows, d_elevs, roi, oht, tht, nsheds, d_obs, d_sheds);

  if (hipMemcpy(h_sheds, d_sheds, nsheds * nr * nwpr * sizeof(ullint), hipMemcpyDeviceToHost)
      != hipSuccess)
    die("hipMemcpy failed");

  if (hipFree(d_elevs) != hipSuccess) die("hipFree failed");
  if (hipFree(d_obs) != hipSuccess) die("hipFree failed");
  if (hipFree(d_sheds) != hipSuccess) die("hipFree failed");
}

////////////////////////////////////////////////////////////////////////////////
// site
////////////////////////////////////////////////////////////////////////////////

__host__ int is_obs_vis(const int cumnwpr,
                        const ullint *cumshed,
                        const int *observer) {
  int i = observer[0] * cumnwpr * 64 + observer[1];
  if ((cumshed[i / 64] & 1ULL << (63 - i % 64)) != 0)
    return 1;
  else
    return 0;
}

__device__ int is_obs_vis(const int cumnwpr,
                          const ullint *cumshed,
                          const int obsx, const int obsy) {
  int i = obsx * cumnwpr * 64 + obsy;
  if ((cumshed[i / 64] & 1ULL << (63 - i % 64)) != 0)
    return 1;
  else
    return 0;
}

__global__ void calc_extra_area(const int nrows, const int roi,
                                const int *obs, const int *updatelist,
                                const ullint *sheds,
                                const int nr, const int nwpr, const int cumnwpr,
                                const ullint *cumshed,
                                int *testshedarea) {
  const int oi = updatelist[blockIdx.x];
  const int obsx = obs[oi*2];
  const int obsy = obs[oi*2+1];
  const ullint *shed = sheds + oi*nr*nwpr;
  int *extraarea = testshedarea + oi;

  extern __shared__ int areas[];
  areas[threadIdx.x] = 0;

  // calculate nrpt rows of extra area
  const int nrpt = nr / blockDim.x + (nr % blockDim.x ? 1 : 0);
  int sum = 0;

  for (int row = threadIdx.x * nrpt; row < (threadIdx.x + 1) * nrpt && row < nr; row++) {
    const int cumrow = obsx - roi + row;
    if (cumrow >= 0 && cumrow < nrows) {
      int firstword = (obsy - roi) / 64;
      int firstbit = (obsy - roi) % 64;
      if (firstbit < 0) {
        firstword--;
        firstbit += 64;
      }
      int lastword = (obsy + roi) / 64;

      ullint prevvalue = 0ULL;
      ullint value, cumvalue, tempvalue;

      for (int cumword = firstword; cumword <= lastword; cumword++)
        if (cumword >= 0 && cumword < cumnwpr) {
          int word = cumword - firstword;  // definition out of loop?
          if (cumword == 0 && word > 0) prevvalue = shed[row * nwpr + word - 1];
          if (word < nwpr)
            value = shed[row * nwpr + word];
          else
            value = 0ULL;
          cumvalue = cumshed[cumrow * cumnwpr + cumword];
          tempvalue = cumvalue;
          tempvalue |= value >> firstbit;
          if (firstbit != 0) tempvalue |= prevvalue << (64 - firstbit);
          tempvalue ^= cumvalue;
          sum += __popcll(tempvalue);
          prevvalue = value;
        }
    }
  }

  areas[threadIdx.x] = sum;
  __syncthreads();  // wait for all threads

  if (threadIdx.x == 0) {
    int sum = 0;
    for (int i = 0; i < blockDim.x; i++)
      sum += areas[i];
    *extraarea = sum;
  }
}

__global__ void union_area(const int nrows, const int roi, const int intervis,
                           const int nsheds, const int *obs,
                           const ullint *sheds,
                           const int nr, const int nwpr, const int cumnwpr,
                           const int nusedsheds, const char *usedq,
                           const int lastobs,
                           const int lastobsx, const int lastobsy,
                           const ullint *cumshed,
                           int *testshedarea, int *updatelist) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int obsx, obsy, valid;

  // calculate multiple extra areas
  for (int oi = tid; oi < nsheds; oi += gridDim.x * blockDim.x) {
    valid = 1;
    if (usedq[oi]) {
      valid = 0;
    } else {
      obsx = obs[oi*2];
      obsy = obs[oi*2 + 1];
      if (nusedsheds > 0) {  // lastobs >= 0
        if (square(obsx-lastobsx) + square(obsy-lastobsy) > square(2*roi))
          valid = 0;
        else if (intervis && !is_obs_vis(cumnwpr, cumshed, obsx, obsy))  // if intervis, reset all after the first
          valid = 0;  // testshedarea[oi] = 0;  // reset invisible ones to zero
      }
    }

    if (valid) {
      /*
      cudaStream_t s;
      cudaStreamCreateWithFlags(&s, cudaStreamNonBlocking);
      calc_extra_area<<<1, nr, nr*sizeof(int), s>>>(nrows, roi, obsx, obsy,
                                                    &sheds[oi*nr*nwpr],
                                                    nr, nwpr, cumnwpr,
                                                    cumshed, &testshedarea[oi]);
      cudaStreamDestroy(s);
      // calc_extra_area<<<1, nr, nr*sizeof(int)>>>(nrows, roi, obsx, obsy, &sheds[oi*nr*nwpr],
      //                                            nr, nwpr, cumnwpr, cumshed, &testshedarea[oi]);
      */
      int index = atomicAdd(updatelist+499999, 1);
      updatelist[index] = oi;
    }
  }
}

__device__ void swap(int *x, int *y) {
  int z = *x;
  *x = *y;
  *y = z;
}

__global__ void findtopobs(const int nsheds, const char *usedq,
                           const int *testshedarea, int *top100) {
  __shared__ int obs[256];
  __shared__ int areas[256];
  obs[threadIdx.x] = 0;
  areas[threadIdx.x] = 0;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int size = gridDim.x * blockDim.x;
  for (int i = tid; i < nsheds; i += size)
    if (!usedq[i]) {
      int extraarea = testshedarea[i];
      if (extraarea > areas[threadIdx.x]) {
        obs[threadIdx.x] = i;
        areas[threadIdx.x] = extraarea;
      }
    }
  __syncthreads();

  if (threadIdx.x < 128 && areas[threadIdx.x + 128] > areas[threadIdx.x]) {
    swap(&obs[threadIdx.x], &obs[threadIdx.x + 128]);
    swap(&areas[threadIdx.x], &areas[threadIdx.x + 128]);
  }
  if (threadIdx.x < 64 && areas[threadIdx.x + 64] > areas[threadIdx.x]) {
    swap(&obs[threadIdx.x], &obs[threadIdx.x + 64]);
    swap(&areas[threadIdx.x], &areas[threadIdx.x + 64]);
  }
  if (threadIdx.x < 32 && areas[threadIdx.x + 32] > areas[threadIdx.x]) {
    swap(&obs[threadIdx.x], &obs[threadIdx.x + 32]);
    swap(&areas[threadIdx.x], &areas[threadIdx.x + 32]);
  }
  if (threadIdx.x < 16 && areas[threadIdx.x + 16] > areas[threadIdx.x]) {
    swap(&obs[threadIdx.x], &obs[threadIdx.x + 16]);
    swap(&areas[threadIdx.x], &areas[threadIdx.x + 16]);
  }
  if (threadIdx.x < 8 && areas[threadIdx.x + 8] > areas[threadIdx.x]) {
    swap(&obs[threadIdx.x], &obs[threadIdx.x + 8]);
    swap(&areas[threadIdx.x], &areas[threadIdx.x + 8]);
  }
  if (threadIdx.x < 4 && areas[threadIdx.x + 4] > areas[threadIdx.x]) {
    swap(&obs[threadIdx.x], &obs[threadIdx.x + 4]);
    swap(&areas[threadIdx.x], &areas[threadIdx.x + 4]);
  }
  if (threadIdx.x < 2 && areas[threadIdx.x + 2] > areas[threadIdx.x]) {
    swap(&obs[threadIdx.x], &obs[threadIdx.x + 2]);
    swap(&areas[threadIdx.x], &areas[threadIdx.x + 2]);
  }
  if (threadIdx.x < 1 && areas[threadIdx.x + 1] > areas[threadIdx.x]) {
    swap(&obs[threadIdx.x], &obs[threadIdx.x + 1]);
    swap(&areas[threadIdx.x], &areas[threadIdx.x + 1]);
  }

  if (threadIdx.x == 0) {
    top100[blockIdx.x] = obs[0];
    top100[blockIdx.x + 100] = areas[0];
  }
}

__global__ void calc_union(const int nrows, const int roi,
                           const int nsheds, const int *obs,
                           const ullint *sheds,
                           const int nr, const int nwpr, const int cumnwpr,
                           const int lastobs, char *usedq,
                           ullint *cumshed) {
  // lastobs >= 0
  __shared__ int lastobsx, lastobsy;
  __shared__ const ullint *shed;
  if (threadIdx.x == 0) {
    usedq[lastobs] = 1;  // set gridDim.x times
    lastobsx = obs[lastobs*2];
    lastobsy = obs[lastobs*2 + 1];
    shed = &sheds[lastobs*nr*nwpr];
  }
  __syncthreads();  // wait for thread 0

  int firstword = (lastobsy - roi) / 64;
  int firstbit = (lastobsy - roi) % 64;
  if (firstbit < 0) {
    firstword--;
    firstbit += 64;
  }

  int row = blockIdx.x*blockDim.x + threadIdx.x;
  if (row < nr) {  // a row of shed
    int cumrow = lastobsx - roi + row;
    if (cumrow >= 0 && cumrow < nrows)  // row inside terrain
      for (int word = 0; word < nwpr; word++) {  // each word of row
        int cumword = firstword + word;
        if (cumword >= 0 && cumword < cumnwpr)  // word inside terrain
          cumshed[cumrow * cumnwpr + cumword] |= shed[row * nwpr + word] >> firstbit;
        if (firstbit != 0 && cumword + 1 >= 0 && cumword + 1 < cumnwpr)  // firstbit != 0 and word + 1 inside terrain
          cumshed[cumrow * cumnwpr + cumword + 1] |= shed[row * nwpr + word] << (64 - firstbit);
      }
  }
}

void site_it(const int nrows, const int roi, const int intervis,
             const int nsheds, const int *h_obs, const ullint *h_sheds, char *selected) {
  const int nr = 2 * roi + 1;
  const int nwpr = (nr + 63) / 64;
  const int cumnwpr = (nrows + 63) / 64;

  int *usedsheds;     // list of sheds used so far.
  char *h_usedq;      // whether each particular shed has been used.
  int h_top100[200];  // top 100 tentative observers and extra areas

  usedsheds = 0;
  h_usedq = 0;
  int *areas = 0;
  usedsheds = new int[nsheds];
  h_usedq = new char[nsheds];
  areas = new int[nsheds];
  if (!usedsheds || !h_usedq || !areas)
    die("Memory exhausted. Program terminates.");
  for (int i = 0; i < nsheds; i++) h_usedq[i] = 0;

  int *d_obs;
  ullint *d_sheds;
  char *d_usedq;
  ullint *d_cumshed;
  int *d_testshedarea;
  int *d_top100;
  if (hipMalloc((void **)&d_obs, 2 * nsheds * sizeof(int)) != hipSuccess)
    die("hipMalloc failed");
  if (hipMalloc((void **)&d_sheds, nsheds * nr * nwpr * sizeof(ullint)) != hipSuccess)
    die("hipMalloc failed");
  if (hipMalloc((void **)&d_usedq, nsheds * sizeof(char)) != hipSuccess)
    die("hipMalloc failed");
  if (hipMalloc((void **)&d_cumshed, nrows * cumnwpr * sizeof(ullint)) != hipSuccess)
    die("hipMalloc failed");
  if (hipMalloc((void **)&d_testshedarea, nsheds * sizeof(int)) != hipSuccess)
    die("hipMalloc failed");
  if (hipMalloc((void **)&d_top100, 200 * sizeof(int)) != hipSuccess)
    die("hipMalloc failed");
  if (hipMemcpy(d_obs, h_obs, 2 * nsheds * sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
    die("hipMemcpy failed");
  if (hipMemcpy(d_sheds, h_sheds, nsheds * nr * nwpr * sizeof(ullint), hipMemcpyHostToDevice)
      != hipSuccess)
    die("hipMemcpy failed");
  if (hipMemset(d_usedq, 0, nsheds * sizeof(char)) != hipSuccess)
    die("hipMemset failed");
  if (hipMemset(d_cumshed, 0, nrows*cumnwpr*sizeof(ullint)) != hipSuccess)
    die("hipMemset failed");

  int *d_updatelist;
  if (hipMalloc((void **)&d_updatelist, 500000*sizeof(int)) != hipSuccess) die("hipMalloc failed");

  int nusedsheds = 0;
  int lastobs = -1;
  int lastobsx = 0;
  int lastobsy = 0;
  int cumarea = 0;
  if (hipMemset(d_testshedarea, 0, nsheds * sizeof(int)) != hipSuccess)
    die("hipMemset failed");
  size_t dimgrid = (nsheds+255)/256;
  size_t dimblock = 256;
  // size_t dimgrid = (nsheds + dimblock - 1) / dimblock;

  //cout << "Total area=" << square(nrows) << endl;
  //cout << "#nusedsheds newshed obsx obsy area extraarea newcumarea areapercentage" << endl;

  while (1) {
    if (hipMemset(d_updatelist, 0, 500000*sizeof(int)) != hipSuccess) die("hipMemset failed");

    union_area<<<dimgrid, dimblock>>>(
        nrows, roi, intervis, nsheds, d_obs, d_sheds, nr, nwpr, cumnwpr,
        nusedsheds, d_usedq, lastobs, lastobsx, lastobsy, d_cumshed, d_testshedarea, d_updatelist);

    int updatelistsize;
    if (hipMemcpy(&updatelistsize, d_updatelist+499999, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) die("hipMemcpy failed");
      //cout << "\nsize = " << updatelistsize;

    calc_extra_area<<<updatelistsize, nr, nr*sizeof(int)>>>(
        nrows, roi, d_obs, d_updatelist, d_sheds, nr, nwpr, cumnwpr, d_cumshed, d_testshedarea);

    hipDeviceSynchronize();

    // find top 100 observers
    findtopobs<<<100, 256>>>(nsheds, d_usedq, d_testshedarea, d_top100);
    if (hipMemcpy(h_top100, d_top100, 200 * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
      die("hipMemcpy failed");

    int newshed = 0;
    int extraarea = 0;
    for (int i = 0; i < 100; i++) {
      if (h_top100[i + 100] > extraarea) {
        extraarea = h_top100[i + 100];
        newshed = h_top100[i];
      }
    }
    if (extraarea == 0) {
      //cout << "No more new observers that will add new area." << endl;
      break;
    }

    usedsheds[nusedsheds++] = newshed;
    h_usedq[newshed] = 1;
    lastobs = newshed;
    lastobsx = h_obs[newshed * 2];
    lastobsy = h_obs[newshed * 2 + 1];

    // set usedq and calculate cumshed
    calc_union<<<nr, 1>>>(nrows, roi, nsheds, d_obs, d_sheds, nr, nwpr, cumnwpr, lastobs, d_usedq, d_cumshed);
    hipDeviceSynchronize();

    cumarea += extraarea;
    double areapercentage = 100.0 * cumarea / square(nrows);
    if (areapercentage > 95)
        break;

    if (nusedsheds == 1) {
      if (hipMemcpy(areas, d_testshedarea, nsheds * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
        die("hipMemcpy failed");
      if (intervis)
        if (hipMemset(d_testshedarea, 0, nsheds * sizeof(int)) != hipSuccess)
          die("hipMemset failed");
    }

    /*
    cout << setw(6) << nusedsheds << setw(6) << newshed
         << setw(6) << h_obs[newshed * 2] << setw(6) << h_obs[newshed * 2 + 1]
         << setw(8) << areas[newshed] << setw(8) << extraarea
         << setw(10) << cumarea << setw(8) << areapercentage << endl;
    */
  }
  cout << " nusedsheds:" << nusedsheds << " coverage:" << 100.0*cumarea/square(nrows);

  if (hipFree(d_obs) != hipSuccess) die("hipFree failed");
  if (hipFree(d_sheds) != hipSuccess) die("hipFree failed");
  if (hipFree(d_usedq) != hipSuccess) die("hipFree failed");
  if (hipFree(d_cumshed) != hipSuccess) die("hipFree failed");
  if (hipFree(d_testshedarea) != hipSuccess) die("hipFree failed");
  if (hipFree(d_top100) != hipSuccess) die("hipFree failed");
  if (hipFree(d_updatelist) != hipSuccess) die("hipFree failed");

  for (int i = 0; i < nsheds; i++)
    selected[i] = h_usedq[i];
  delete[] usedsheds;
  delete[] h_usedq;
  delete[] areas;
}

////////////////////////////////////////////////////////////////////////////////
// main
////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv) {
  // clock_t begin, middle, end;
  double elapsed_secs;
  // begin = middle = clock();
  struct timeval begin, middle, end;
  gettimeofday(&begin, NULL);
  middle = begin;

  int nrows;                      // # rows, cols in this cell
  int roi;                        // radius of interest
  int oht;                        // ht of observer above terrain
  int tht;                        // target ht above terrain
  int ntests;                     // # of targets tested per observer
  int blocksize0;                 // Requested number of rows in one block of the input visibility index array.
  float blocksize;               // Perturbed blocksize0, to remove sliver blocks at the end.
  int nwanted0;                   // Desired number of output observers.
  int nwanted;                    // Modified because blocksize was perturbed.
  int nblocks;                    // Number of blocks on one side of it.
  int nwantedperblock;            // Number of observers to find per block, before culling.
  int intervis;                   // Should new observers be visible to existing ones?

  usint *elevs;      // terrain elevation (input)
  unsigned char *vix;             // visibility index * 256 (output); vim
  int *obs;                       // observers
  ullint *sheds;  // viewsheds
  char *selected;

  //cerr << "[SITE, compiled from " << __FILE__ << " on " << __DATE__ << ", " << __TIME__ << ']' << endl;
  if (argc != 10) {
    cerr << "argc=" << argc << endl;
    die("SITE requires 9 arguments: nrows, roi, oht/tht, ntests, blocksize, nwanted, intervis, infile, outfile");
  }

  nrows = atoi(argv[1]);
  roi = atoi(argv[2]);
  oht = tht = atoi(argv[3]);
  ntests = atoi(argv[4]);
  blocksize0 = atoi(argv[5]);
  nwanted0 = atoi(argv[6]);
  intervis = atoi(argv[7]);

  //cerr << "nrows=" << nrows << ", roi=" << roi << ", oht=" << oht << ", tht=" << tht
  //     << ", ntests=" << ntests << "\nblocksize0=" << blocksize0 << ", nwanted0=" << nwanted0
  //     << ", intervis=" << intervis << endl;

  if (nrows <= 0 || nrows > 20000) die("Unreasonable value for nrows");
  if (roi < 1 || roi > 10000) die("Unreasonable value for roi.");
  if (tht < 0 || tht > 1000000) die("Unreasonable value for tht.");
  if (ntests < 1 || ntests > 1000) die("Unreasonable value for ntests.");
  if (blocksize0 < 10 || blocksize0 > 2000) die("Unreasonable value for blocksize0.");
  if (nwanted0 < 100 || nwanted0 > 2000000) die("Unreasonable value for nwanted0.");
  if (intervis != 0 && intervis != 1) die("Unreasonable value for intervis.");

  // Perturb blocksize so that the last block won't be really small.
  blocksize = (float)nrows / (int)((float)nrows / blocksize0 + 0.5f);  // floating point block size
  nblocks = (int)(nrows / blocksize + 0.5f);                             // number of blocks
  nwantedperblock = (int)((float)nwanted0 / square(nblocks) + 0.99999f);  // number of wanted per block
  nwanted = nwantedperblock * square(nblocks);                          // number of wanted
  int lastsize = nrows - (int)(blocksize * (nblocks - 1));              // size of the last block
  if (square(lastsize) < nwantedperblock)                               // too small
    die("The last block is too small for nwantedperblock.");

  //cerr << "blocksize=" << blocksize << ", nblocks=" << nblocks
  //     << ", nwantedperblock=" << nwantedperblock << ", nwanted=" << nwanted << endl;

  // number of rows per shed and number of words per row
  const int nr = 2 * roi + 1;
  const int nwpr = (nr + 63) / 64;

  elevs = 0;
  vix = 0;
  obs = 0;
  sheds = 0;
  selected = 0;
  elevs = new usint[square(nrows)];
  vix = new unsigned char[square(nrows)];
  obs = new int[2 * nwanted];
  sheds = new ullint[nwanted * nr * nwpr];
  selected = new char[nwanted];
  if (!elevs || !vix || !obs || !sheds || !selected)
    die("Memory exhausted. Program terminates.");
/*
  if (cudaMallocHost((void **)&elevs, square(nrows) * sizeof(usint)) != cudaSuccess)
    die("cudaMallocHost failed");
  if (cudaMallocHost((void **)&vix, square(nrows) * sizeof(unsigned char)) != cudaSuccess)
    die("cudaMallocHost failed");
  if (cudaMallocHost((void **)&obs, 2 * nwanted * sizeof(int)) != cudaSuccess)
    die("cudaMallocHost failed");
  if (cudaMallocHost((void **)&sheds, nwanted * nr * nwpr * sizeof(ullint)) != cudaSuccess)
    die("cudaMallocHost failed");
*/
  /*
  for (int i = 0; i < nrows; i++)
    for (int j = 0; j < nrows; j++) {
      cin.read(reinterpret_cast<char *>(&elevs[i * nrows + j]), sizeof(usint));
      if (cin.fail()) {
        cerr << "Error: at i=" << i << ", j=" << j << endl;
        die("Input failed");
      }
    }
  */
  ifstream ifs(argv[8]);
  ifs.read((char *)elevs, square(nrows)*sizeof(usint));
  if (ifs.fail())
    die("Input failed");
  ifs.close();

  // end = clock();
  // elapsed_secs = float(end - middle) / CLOCKS_PER_SEC;
  gettimeofday(&end, NULL);
  elapsed_secs = calc_time(middle, end);
  middle = end;
  cerr << "input:" << elapsed_secs;

  calc_vix(nrows, elevs, roi, oht, tht, ntests, vix);
  hipDeviceSynchronize();
  // end = clock();
  // elapsed_secs = float(end - middle) / CLOCKS_PER_SEC;
  gettimeofday(&end, NULL);
  elapsed_secs = calc_time(middle, end);
  middle = end;
  cerr << " vix:" << elapsed_secs;
  /*
  int x;
  ullint sum = 0;
  ifstream ifs("vim.bin");
  for (int i = 0; i < square(nrows); i++) {
    ifs >> x;
    sum += square(vix[i]-x);
  }
  cerr << " RMS VIM error:" << sqrt((double)sum/square(nrows));
  ifs.close();
  */

  find_max(nrows, vix, blocksize, nwanted, nblocks, nwantedperblock, obs);
  hipDeviceSynchronize();
  // end = clock();
  // elapsed_secs = float(end - middle) / CLOCKS_PER_SEC;
  gettimeofday(&end, NULL);
  elapsed_secs = calc_time(middle, end);
  middle = end;
  cerr << " findmax:" << elapsed_secs;
  
  calc_sheds(nrows, elevs, roi, oht, tht, nwanted, obs, sheds);
  hipDeviceSynchronize();
  // end = clock();
  // elapsed_secs = float(end - middle) / CLOCKS_PER_SEC;
  gettimeofday(&end, NULL);
  elapsed_secs = calc_time(middle, end);
  middle = end;
  cerr << " viewshed:" << elapsed_secs;

  site_it(nrows, roi, intervis, nwanted, obs, sheds, selected);
  hipDeviceSynchronize();
  // end = clock();
  // elapsed_secs = float(end - middle) / CLOCKS_PER_SEC;
  gettimeofday(&end, NULL);
  elapsed_secs = calc_time(middle, end);
  middle = end;
  cerr << " site:" << elapsed_secs;

  ofstream ofs(argv[9]);
  for (int i = 0; i < nwanted; i++)
    if (selected[i])
      ofs << obs[2*i] << ',' << obs[2*i+1] << '\n';
  ofs.close();

  delete[] elevs;
  delete[] vix;
  delete[] obs;
  delete[] sheds;
  delete[] selected;
/*
  if (cudaFreeHost(elevs) != cudaSuccess) die("cudaFreeHost failed");
  if (cudaFreeHost(vix) != cudaSuccess) die("cudaFreeHost failed");
  if (cudaFreeHost(obs) != cudaSuccess) die("cudaFreeHost failed");
  if (cudaFreeHost(sheds) != cudaSuccess) die("cudaFreeHost failed");
*/
  gettimeofday(&end, NULL);
  elapsed_secs = calc_time(middle, end);
  middle = end;
  cerr << " output:" << elapsed_secs;
  // end = clock();
  // elapsed_secs = float(end - begin) / CLOCKS_PER_SEC;
  gettimeofday(&end, NULL);
  elapsed_secs = calc_time(begin, end);
  cerr << " total:" << elapsed_secs << endl;
}
